#include "hip/hip_runtime.h"
	
	#include <omp.h>
	#include <openacc.h>
	#include <stdlib.h>
	#include <math.h>
	#include <array>
	#include <vector>
	#include <sstream>
	#include <chrono>
	#include <random>
	#include <limits>
	#include <memory>
	#include <cstddef>
	#include <type_traits>
	//#include <hip/hip_runtime.h>
	//#include <openacc_curand.h>
	#include <hiprand/hiprand_kernel.h>
	
	#include "../include/musket.hpp"
	#include "../include/nbody-n-1-g-4_0.hpp"
		
			
	const int dim = 500000;
	const int steps = 5;
	const float EPSILON = 1.0E-10f;
	const float DT = 0.01f;
	mkt::DArray<Particle> P(0, 500000, 500000, Particle{}, 1, 0, 0, mkt::DIST, mkt::DIST);
	mkt::DArray<Particle> oldP(0, 500000, 500000, Particle{}, 1, 0, 0, mkt::COPY, mkt::COPY);
	
	//Particle::Particle() : x(), y(), z(), vx(), vy(), vz(), mass(), charge() {}
	
	// __global__ void setup_kernel(hiprandState *state)
	// {
	// 	int id = threadIdx.x + blockIdx.x * 1024;
	// 	/* Each thread gets same seed, a different sequence 
	// 	   number, no offset */
	// 	hiprand_init(1234, id, 0, &state[id]);
	// }
	
	struct Init_particles_map_index_in_place_array_functor{
		
		Init_particles_map_index_in_place_array_functor(){
			printf("init functor constructor\n");
		}
		
		~Init_particles_map_index_in_place_array_functor() {}
		
		__device__
		auto operator()(int i, Particle& p){
			hiprandState state;
			hiprand_init(clock64(), 0, 0, &state);
			
			p.x = static_cast<float>(hiprand_uniform(&state) * (1.0f - 0.0f + 0.999999) + 0.0f);
			p.y = static_cast<float>(hiprand_uniform(&state) * (1.0f - 0.0f + 0.999999) + 0.0f);
			p.z = static_cast<float>(hiprand_uniform(&state) * (1.0f - 0.0f + 0.999999) + 0.0f);
			p.vx = 0.0f;
			p.vy = 0.0f;
			p.vz = 0.0f;
			p.mass = 1.0f;
			p.charge = (1.0f - (2.0f * static_cast<float>(((i) % 2))));

		}
	
		void init(int gpu){

		}
	};
	struct Calc_force_map_index_in_place_array_functor{
		
		Calc_force_map_index_in_place_array_functor(const mkt::DArray<Particle>& _oldP) : oldP(_oldP){
			printf("functor constructor \n");
		}
		
		~Calc_force_map_index_in_place_array_functor() {}
		
		__device__
		auto operator()(int curIndex, Particle& curParticle){
			float ax = 0.0f;
			float ay = 0.0f;
			float az = 0.0f;
			for(int j = 0; ((j) < 500000); j++){
				
				if(((j) != (curIndex))){
				float dx;
				float dy;
				float dz;
				float r2;
				float r;
				float qj_by_r3;
				dx = ((curParticle).x - oldP.get_data_local((j)).x);
				dy = ((curParticle).y - oldP.get_data_local((j)).y);
				dz = ((curParticle).z - oldP.get_data_local((j)).z);
				r2 = ((((dx) * (dx)) + ((dy) * (dy))) + ((dz) * (dz)));
				r = sqrtf((r2));
				
				if(((r) < (EPSILON))){
				qj_by_r3 = 0.0f;
				}
				 else {
						qj_by_r3 = (oldP.get_data_local((j)).charge / ((r2) * (r)));
					}
				ax += ((qj_by_r3) * (dx));
				ay += ((qj_by_r3) * (dy));
				az += ((qj_by_r3) * (dz));
				}
			}
			float vx0 = (curParticle).vx;
			float vy0 = (curParticle).vy;
			float vz0 = (curParticle).vz;
			float qidt_by_m = (((curParticle).charge * (DT)) / (curParticle).mass);
			curParticle.vx += ((ax) * (qidt_by_m));
			curParticle.vy += ((ay) * (qidt_by_m));
			curParticle.vz += ((az) * (qidt_by_m));
			curParticle.x += ((((vx0) + (curParticle).vx) * (DT)) * 0.5f);
			curParticle.y += ((((vy0) + (curParticle).vy) * (DT)) * 0.5f);
			curParticle.z += ((((vz0) + (curParticle).vz) * (DT)) * 0.5f);
		}
	
		void init(int gpu){
			printf("functor init %i\n", gpu)
			oldP.init(gpu);
		}
			
		
		mkt::DeviceArray<Particle> oldP;

	};
	
	
	
	
	int main(int argc, char** argv) {
		
		//hiprandState* devStates[4];
		
		// #pragma omp parallel for
		// for(int gpu = 0; gpu < 4; ++gpu){
		// 	hipSetDevice(gpu);
		// 	hipMalloc((void **)&devStates[gpu], 64 * 1024 * sizeof(hiprandState)));
		// 	setup_kernel<<<64, 1024>>>(devStates[gpu]);
		// }
		
		Init_particles_map_index_in_place_array_functor init_particles_map_index_in_place_array_functor{};
		Calc_force_map_index_in_place_array_functor calc_force_map_index_in_place_array_functor{oldP};
		
				
		printf("map init\n");
		mkt::map_index_in_place<Particle, Init_particles_map_index_in_place_array_functor>(P, init_particles_map_index_in_place_array_functor);
		printf("gather init\n");
		mkt::gather<Particle>(P, oldP);
		mkt::sync_streams();

		double gather_time = 0.0;
		double map_time = 0.0;

		std::chrono::high_resolution_clock::time_point timer_start = std::chrono::high_resolution_clock::now();
		for(int i = 0; ((i) < (steps)); ++i){
			mkt::sync_streams();
			std::chrono::high_resolution_clock::time_point map_timer_start = std::chrono::high_resolution_clock::now();
			printf("map iteration %i\n", i);
			mkt::map_index_in_place<Particle, Calc_force_map_index_in_place_array_functor>(P, calc_force_map_index_in_place_array_functor);

			mkt::sync_streams();
			printf("map end iteration %i\n", i);
			std::chrono::high_resolution_clock::time_point map_timer_end = std::chrono::high_resolution_clock::now();

			map_time += std::chrono::duration<double>(map_timer_end - map_timer_start).count();

			std::chrono::high_resolution_clock::time_point gather_timer_start = std::chrono::high_resolution_clock::now();

			printf("gather iteration %i\n", i);
			mkt::gather<Particle>(P, oldP);
			mkt::sync_streams();
			printf("gather iteration %i\n", i);

			std::chrono::high_resolution_clock::time_point gather_timer_end = std::chrono::high_resolution_clock::now();

			gather_time += std::chrono::duration<double>(gather_timer_end - gather_timer_start).count();
		}
		mkt::sync_streams();
		std::chrono::high_resolution_clock::time_point timer_end = std::chrono::high_resolution_clock::now();
		double seconds = std::chrono::duration<double>(timer_end - timer_start).count();
		
		printf("Execution time: %.5fs\n", seconds);

		printf("map time: %.5fs\n", map_time);
		printf("gather time: %.5fs\n", gather_time);

		printf("Threads: %i\n", omp_get_max_threads());
		printf("Processes: %i\n", 1);
		
		// #pragma omp parallel for
		// for(int gpu = 0; gpu < 4; ++gpu){
		// 	hipSetDevice(gpu);
		// 	hipFree(devStates[gpu]);
		// }
		return EXIT_SUCCESS;
		}
