#include "hip/hip_runtime.h"
	#include <mpi.h>
	#include <hip/hip_runtime.h>
	#include <omp.h>
	#include <stdlib.h>
	#include <math.h>
	#include <array>
	#include <vector>
	#include <sstream>
	#include <chrono>
	#include <hiprand/hiprand_kernel.h>
	#include <limits>
	#include <memory>
	#include <cstddef>
	#include <type_traits>
	
	
	#include "../include/musket.cuh"
	#include "../include/fss-n-16-g-4_12.cuh"
	
	const size_t number_of_processes = 16;
	const size_t process_id = 12;
	int mpi_rank = -1;
	int mpi_world_size = 0;
	
	
	const double PI = 3.141592653589793;
	const double EULER = 2.718281828459045;
	const double UPPER_BOUND = 5.12;
	const double LOWER_BOUND = -5.12;
	const double PROBLEM_RANGE = 10.24;
	const double INIT_UPPER_BOUND = 5.12;
	const double INIT_LOWER_BOUND = -5.12;
	const double WEIGHT_UPPER_BOUND = 5000.0;
	const double WEIGHT_LOWER_BOUND = 1.0;
	const double STEP_SIZE_INITIAL = 0.1;
	const double STEP_SIZE_FINAL = 1.0E-5;
	const double STEP_SIZE_VOLITIVE_INITIAL = 0.2;
	const double STEP_SIZE_VOLITIVE_FINAL = 2.0E-5;
	const int NUMBER_OF_FISH = 2048;
	const int ITERATIONS = 5000;
	const int DIMENSIONS = 512;
	
	//Fish::Fish() : position(0, 0.0), fitness(), candidate_position(0, 0.0), candidate_fitness(), displacement(0, 0.0), fitness_variation(), weight(), best_position(0, 0.0), best_fitness() {}
	

	
	struct InitFish_map_in_place_array_functor{
		
		InitFish_map_in_place_array_functor(){}
		
		~InitFish_map_in_place_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			hiprandState_t curand_state; // performance could be improved by creating states before
			size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			hiprand_init(clock64(), id, 0, &curand_state);
			fi.fitness = std::numeric_limits<double>::lowest();
			fi.candidate_fitness = std::numeric_limits<double>::lowest();
			fi.weight = (WEIGHT_LOWER_BOUND);
			fi.fitness_variation = 0.0;
			fi.best_fitness = std::numeric_limits<double>::lowest();
			for(int i = 0; ((i) < (DIMENSIONS)); ++i){
				fi.position[(i)] = static_cast<double>(hiprand_uniform(&curand_state) * ((INIT_UPPER_BOUND) - (INIT_LOWER_BOUND)) + (INIT_LOWER_BOUND));
				fi.candidate_position[(i)] = 0.0;
				fi.displacement[(i)] = 0.0;
				fi.best_position[(i)] = 0.0;
			}
			return (fi);
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
	};
	struct EvaluateFitness_map_in_place_array_functor{
		
		EvaluateFitness_map_in_place_array_functor(){}
		
		~EvaluateFitness_map_in_place_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			double sum = 0.0;
			for(int j = 0; ((j) < (DIMENSIONS)); ++j){
				double value = (fi).position[(j)];
				sum += (std::pow((value), 2) - (10 * std::cos(((2 * (PI)) * (value)))));
			}
			fi.fitness = -(((10 * (DIMENSIONS)) + (sum)));
			
			if(((fi).fitness > (fi).best_fitness)){
			fi.best_fitness = (fi).fitness;
			for(int k = 0; ((k) < (DIMENSIONS)); ++k){
				fi.best_position[(k)] = (fi).position[(k)];
			}
			}
			return (fi);
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
	};
	struct IndividualMovement_map_in_place_array_functor{
		
		IndividualMovement_map_in_place_array_functor(){}
		
		~IndividualMovement_map_in_place_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			hiprandState_t curand_state; // performance could be improved by creating states before
			size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			hiprand_init(clock64(), id, 0, &curand_state);
			for(int i = 0; ((i) < (DIMENSIONS)); ++i){
				double rand_factor = static_cast<double>(hiprand_uniform(&curand_state) * (1.0 - -(1.0)) + -(1.0));
				double direction = (((rand_factor) * (step_size)) * ((UPPER_BOUND) - (LOWER_BOUND)));
				double new_value = ((fi).position[(i)] + (direction));
				
				if(((new_value) < (LOWER_BOUND))){
				new_value = (LOWER_BOUND);
				} else 
				if(((new_value) > (UPPER_BOUND))){
				new_value = (UPPER_BOUND);
				}
				fi.candidate_position[(i)] = (new_value);
			}
			double sum = 0.0;
			for(int j = 0; ((j) < (DIMENSIONS)); ++j){
				double value = (fi).candidate_position[(j)];
				sum += (std::pow((value), 2) - (10 * std::cos(((2 * (PI)) * (value)))));
			}
			fi.candidate_fitness = -(((10 * (DIMENSIONS)) + (sum)));
			
			if(((fi).candidate_fitness > (fi).fitness)){
			fi.fitness_variation = ((fi).candidate_fitness - (fi).fitness);
			fi.fitness = (fi).candidate_fitness;
			for(int k = 0; ((k) < (DIMENSIONS)); ++k){
				fi.displacement[(k)] = ((fi).candidate_position[(k)] - (fi).position[(k)]);
				fi.position[(k)] = (fi).candidate_position[(k)];
			}
			
			if(((fi).fitness > (fi).best_fitness)){
			fi.best_fitness = (fi).fitness;
			for(int k = 0; ((k) < (DIMENSIONS)); ++k){
				fi.best_position[(k)] = (fi).position[(k)];
			}
			}
			}
			 else {
					fi.fitness_variation = 0.0;
					for(int k = 0; ((k) < (DIMENSIONS)); ++k){
						fi.displacement[(k)] = 0.0;
					}
				}
			return (fi);
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		double step_size;
		
	};
	struct Feeding_map_in_place_array_functor{
		
		Feeding_map_in_place_array_functor(){}
		
		~Feeding_map_in_place_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			
			if(((max_fitness_variation) != 0.0)){
			double result = ((fi).weight + ((fi).fitness_variation / (max_fitness_variation)));
			
			if(((result) > (WEIGHT_UPPER_BOUND))){
			result = (WEIGHT_UPPER_BOUND);
			} else 
			if(((result) < (WEIGHT_LOWER_BOUND))){
			result = (WEIGHT_LOWER_BOUND);
			}
			fi.weight = (result);
			}
			return (fi);
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		double max_fitness_variation;
		
	};
	struct CalcDisplacementMap_map_in_place_array_functor{
		
		CalcDisplacementMap_map_in_place_array_functor(){}
		
		~CalcDisplacementMap_map_in_place_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			for(int i = 0; ((i) < (DIMENSIONS)); ++i){
				fi.displacement[(i)] *= (fi).fitness_variation;
			}
			return (fi);
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
	};
	struct CalcInstinctiveMovementVector_map_in_place_array_functor{
		
		CalcInstinctiveMovementVector_map_in_place_array_functor(){}
		
		~CalcInstinctiveMovementVector_map_in_place_array_functor() {}
		
		__device__
		auto operator()(double x){
			double result = (x);
			
			if(((sum_fitness_variation) != 0.0)){
			result = ((x) / (sum_fitness_variation));
			}
			x = (result);
			return (x);
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		double sum_fitness_variation;
		
	};
	struct InstinctiveMovement_map_in_place_array_functor{
		
		InstinctiveMovement_map_in_place_array_functor(const mkt::DArray<double>& _instinctive_movement_vector_copy) : instinctive_movement_vector_copy(_instinctive_movement_vector_copy){}
		
		~InstinctiveMovement_map_in_place_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			for(int i = 0; ((i) < (DIMENSIONS)); ++i){
				double new_position = ((fi).position[(i)] + instinctive_movement_vector_copy.get_data_local((i)));
				
				if(((new_position) < (LOWER_BOUND))){
				new_position = (LOWER_BOUND);
				} else 
				if(((new_position) > (UPPER_BOUND))){
				new_position = (UPPER_BOUND);
				}
				fi.position[(i)] = (new_position);
			}
			return (fi);
		}
	
		void init(int device){
			instinctive_movement_vector_copy.init(device);
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
		mkt::DeviceArray<double> instinctive_movement_vector_copy;
	};
	struct CalcWeightedFish_map_array_functor{
		
		CalcWeightedFish_map_array_functor(){}
		
		~CalcWeightedFish_map_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			for(int i = 0; ((i) < (DIMENSIONS)); ++i){
				fi.position[(i)] *= (fi).weight;
			}
			return (fi);
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
	};
	struct CalcBarycenterMap_map_in_place_array_functor{
		
		CalcBarycenterMap_map_in_place_array_functor(){}
		
		~CalcBarycenterMap_map_in_place_array_functor() {}
		
		__device__
		auto operator()(double x){
			double result = (x);
			
			if(((sum_weight) != 0)){
			result = ((x) / (sum_weight));
			}
			x = (result);
			return (x);
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		double sum_weight;
		
	};
	struct VolitiveMovement_map_in_place_array_functor{
		
		VolitiveMovement_map_in_place_array_functor(const mkt::DArray<double>& _barycenter_copy) : barycenter_copy(_barycenter_copy){}
		
		~VolitiveMovement_map_in_place_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			hiprandState_t curand_state; // performance could be improved by creating states before
			size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			hiprand_init(clock64(), id, 0, &curand_state);
			double distance = 0.0;
			for(int i = 0; ((i) < (DIMENSIONS)); ++i){
				distance += (((fi).position[(i)] - barycenter_copy.get_data_local((i))) * ((fi).position[(i)] - barycenter_copy.get_data_local((i))));
			}
			distance = sqrt((distance));
			
			if(((distance) != 0.0)){
			double rand_factor = static_cast<double>(hiprand_uniform(&curand_state) * (1.0 - 0.0) + 0.0);
			for(int i = 0; ((i) < (DIMENSIONS)); ++i){
				double direction = ((((rand_factor) * (step_size)) * ((UPPER_BOUND) - (LOWER_BOUND))) * (((fi).position[(i)] - barycenter_copy.get_data_local((i))) / (distance)));
				double new_position = (fi).position[(i)];
				
				if(((sum_weight) > (sum_weight_last_iteration))){
				new_position -= (direction);
				}
				 else {
						new_position += (direction);
					}
				
				if(((new_position) < (LOWER_BOUND))){
				new_position = (LOWER_BOUND);
				} else 
				if(((new_position) > (UPPER_BOUND))){
				new_position = (UPPER_BOUND);
				}
				fi.position[(i)] = (new_position);
			}
			}
			return (fi);
		}
	
		void init(int device){
			barycenter_copy.init(device);
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		double step_size;
		double sum_weight;
		double sum_weight_last_iteration;
		
		mkt::DeviceArray<double> barycenter_copy;
	};
	struct Lambda56_map_reduce_array_functor{
		
		Lambda56_map_reduce_array_functor(){}
		
		~Lambda56_map_reduce_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			return (fi).weight;
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
	};
	struct Lambda57_map_reduce_array_functor{
		
		Lambda57_map_reduce_array_functor(){}
		
		~Lambda57_map_reduce_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			return (fi).fitness_variation;
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
	};
	struct Lambda58_map_reduce_array_functor{
		
		Lambda58_map_reduce_array_functor(){}
		
		~Lambda58_map_reduce_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			return (fi).fitness_variation;
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
	};
	struct Lambda59_map_reduce_array_functor{
		
		Lambda59_map_reduce_array_functor(){}
		
		~Lambda59_map_reduce_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			return (fi).displacement;
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
	};
	struct Lambda60_map_reduce_array_functor{
		
		Lambda60_map_reduce_array_functor(){}
		
		~Lambda60_map_reduce_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			return (fi).weight;
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
	};
	struct Lambda61_map_reduce_array_functor{
		
		Lambda61_map_reduce_array_functor(){}
		
		~Lambda61_map_reduce_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			return (fi).position;
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
	};
	struct Lambda62_map_reduce_array_functor{
		
		Lambda62_map_reduce_array_functor(){}
		
		~Lambda62_map_reduce_array_functor() {}
		
		__device__
		auto operator()(Fish fi){
			return (fi).best_fitness;
		}
	
		void init(int device){
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
	};
	
	
	
	
	template<>
	double mkt::map_reduce_plus<Fish, double, Lambda56_map_reduce_array_functor>(mkt::DArray<Fish>& a, Lambda56_map_reduce_array_functor f){
			double local_result = 0.0;
			double global_result = 0.0;
						
			if(a.get_device_distribution() == mkt::Distribution::DIST){
				std::array<double*,4> d_odata;
				std::array<double, 4> gpu_results;
				const int gpu_elements = a.get_size_gpu();
				int threads = gpu_elements < 1024 ? gpu_elements : 1024; // nextPow2
				int blocks = (gpu_elements + threads - 1) / threads;
		
				for(int gpu = 0; gpu < 4; ++gpu){
					hipSetDevice(gpu);
					hipMalloc((void**) &d_odata[gpu], blocks * sizeof(double));
					Fish* devptr = a.get_device_pointer(gpu);
					
					mkt::kernel::map_reduce_plus_call<Fish, double, Lambda56_map_reduce_array_functor>(gpu_elements, devptr, d_odata[gpu], threads, blocks, f, mkt::cuda_streams[gpu], gpu);
				}
				mkt::sync_streams();
				
				// fold on gpus: step 2
				while(blocks > 1){
			      int threads_2 = blocks < 1024 ? blocks : 1024; // nextPow2
			      int blocks_2 = (blocks + threads_2 - 1) / threads_2;
				  for(int gpu = 0; gpu < 4; ++gpu){
				      hipSetDevice(gpu);
				      mkt::kernel::reduce_plus_call<double>(blocks, d_odata[gpu], d_odata[gpu], threads_2, blocks_2, mkt::cuda_streams[gpu], gpu);
				  }
				  blocks = blocks_2;
			  	  mkt::sync_streams();
			  	}
				
				// copy final sum from device to host
				  for (int gpu = 0; gpu < 4; ++gpu) {
				    hipSetDevice(gpu);
				    hipMemcpyAsync(&gpu_results[gpu], d_odata[gpu], sizeof(double), hipMemcpyDeviceToHost, mkt::cuda_streams[gpu]);
				  }
				  mkt::sync_streams();
				  
				  for(int gpu = 0; gpu < 4; ++gpu) {
					hipSetDevice(gpu);
					hipFree(d_odata[gpu]);
				  }
				
				for(int gpu = 0; gpu < 4; ++gpu){
					local_result = local_result + gpu_results[gpu];
				}
			}else if(a.get_device_distribution() == mkt::Distribution::COPY){ // use only gpu 0, since all have the same data
				const int gpu_elements = a.get_size_gpu();
				int threads = gpu_elements < 1024 ? gpu_elements : 1024; // nextPow2
				int blocks = (gpu_elements + threads - 1) / threads;
				hipSetDevice(0);
				double* d_odata;
				hipMalloc((void**) &d_odata, blocks * sizeof(double));
				Fish* devptr = a.get_device_pointer(0);
				
				mkt::kernel::map_reduce_plus_call<Fish, double, Lambda56_map_reduce_array_functor>(gpu_elements, devptr, d_odata, threads, blocks, f, mkt::cuda_streams[0], 0);
				
				// fold on gpus: step 2
				while(blocks > 1){
				  int threads_2 = blocks < 1024 ? blocks : 1024; // nextPow2
				  int blocks_2 = (blocks + threads_2 - 1) / threads_2;
				  mkt::kernel::reduce_plus_call<double>(blocks, d_odata, d_odata, threads_2, blocks_2, mkt::cuda_streams[0], 0);
				  blocks = blocks_2;
				}
				
				// copy final sum from device to host
				  hipMemcpyAsync(&local_result, d_odata, sizeof(double), hipMemcpyDeviceToHost, mkt::cuda_streams[0]);
				  mkt::sync_streams();
				hipFree(d_odata);
			}
			
			if(a.get_distribution() == mkt::Distribution::DIST){
				MPI_Allreduce(&local_result, &global_result, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
				return global_result;
			}else if(a.get_distribution() == mkt::Distribution::COPY){
				return local_result;
			}
	}
	template<>
	double mkt::map_reduce_max<Fish, double, Lambda57_map_reduce_array_functor>(mkt::DArray<Fish>& a, Lambda57_map_reduce_array_functor f){
			double local_result = std::numeric_limits<double>::lowest();
			double global_result = std::numeric_limits<double>::lowest();
						
			if(a.get_device_distribution() == mkt::Distribution::DIST){
				std::array<double*,4> d_odata;
				std::array<double, 4> gpu_results;
				const int gpu_elements = a.get_size_gpu();
				int threads = gpu_elements < 1024 ? gpu_elements : 1024; // nextPow2
				int blocks = (gpu_elements + threads - 1) / threads;
		
				for(int gpu = 0; gpu < 4; ++gpu){
					hipSetDevice(gpu);
					hipMalloc((void**) &d_odata[gpu], blocks * sizeof(double));
					Fish* devptr = a.get_device_pointer(gpu);
					
					mkt::kernel::map_reduce_max_call<Fish, double, Lambda57_map_reduce_array_functor>(gpu_elements, devptr, d_odata[gpu], threads, blocks, f, mkt::cuda_streams[gpu], gpu);
				}
				mkt::sync_streams();
				
				// fold on gpus: step 2
				while(blocks > 1){
			      int threads_2 = blocks < 1024 ? blocks : 1024; // nextPow2
			      int blocks_2 = (blocks + threads_2 - 1) / threads_2;
				  for(int gpu = 0; gpu < 4; ++gpu){
				      hipSetDevice(gpu);
				      mkt::kernel::reduce_max_call<double>(blocks, d_odata[gpu], d_odata[gpu], threads_2, blocks_2, mkt::cuda_streams[gpu], gpu);
				  }
				  blocks = blocks_2;
			  	  mkt::sync_streams();
			  	}
				
				// copy final sum from device to host
				  for (int gpu = 0; gpu < 4; ++gpu) {
				    hipSetDevice(gpu);
				    hipMemcpyAsync(&gpu_results[gpu], d_odata[gpu], sizeof(double), hipMemcpyDeviceToHost, mkt::cuda_streams[gpu]);
				  }
				  mkt::sync_streams();
				  
				  for(int gpu = 0; gpu < 4; ++gpu) {
					hipSetDevice(gpu);
					hipFree(d_odata[gpu]);
				  }
				
				for(int gpu = 0; gpu < 4; ++gpu){
					local_result = local_result > gpu_results[gpu] ? local_result : gpu_results[gpu];
				}
			}else if(a.get_device_distribution() == mkt::Distribution::COPY){ // use only gpu 0, since all have the same data
				const int gpu_elements = a.get_size_gpu();
				int threads = gpu_elements < 1024 ? gpu_elements : 1024; // nextPow2
				int blocks = (gpu_elements + threads - 1) / threads;
				hipSetDevice(0);
				double* d_odata;
				hipMalloc((void**) &d_odata, blocks * sizeof(double));
				Fish* devptr = a.get_device_pointer(0);
				
				mkt::kernel::map_reduce_max_call<Fish, double, Lambda57_map_reduce_array_functor>(gpu_elements, devptr, d_odata, threads, blocks, f, mkt::cuda_streams[0], 0);
				
				// fold on gpus: step 2
				while(blocks > 1){
				  int threads_2 = blocks < 1024 ? blocks : 1024; // nextPow2
				  int blocks_2 = (blocks + threads_2 - 1) / threads_2;
				  mkt::kernel::reduce_max_call<double>(blocks, d_odata, d_odata, threads_2, blocks_2, mkt::cuda_streams[0], 0);
				  blocks = blocks_2;
				}
				
				// copy final sum from device to host
				  hipMemcpyAsync(&local_result, d_odata, sizeof(double), hipMemcpyDeviceToHost, mkt::cuda_streams[0]);
				  mkt::sync_streams();
				hipFree(d_odata);
			}
			
			if(a.get_distribution() == mkt::Distribution::DIST){
				MPI_Allreduce(&local_result, &global_result, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
				return global_result;
			}else if(a.get_distribution() == mkt::Distribution::COPY){
				return local_result;
			}
	}
	template<>
	double mkt::map_reduce_plus<Fish, double, Lambda58_map_reduce_array_functor>(mkt::DArray<Fish>& a, Lambda58_map_reduce_array_functor f){
			double local_result = 0.0;
			double global_result = 0.0;
						
			if(a.get_device_distribution() == mkt::Distribution::DIST){
				std::array<double*,4> d_odata;
				std::array<double, 4> gpu_results;
				const int gpu_elements = a.get_size_gpu();
				int threads = gpu_elements < 1024 ? gpu_elements : 1024; // nextPow2
				int blocks = (gpu_elements + threads - 1) / threads;
		
				for(int gpu = 0; gpu < 4; ++gpu){
					hipSetDevice(gpu);
					hipMalloc((void**) &d_odata[gpu], blocks * sizeof(double));
					Fish* devptr = a.get_device_pointer(gpu);
					
					mkt::kernel::map_reduce_plus_call<Fish, double, Lambda58_map_reduce_array_functor>(gpu_elements, devptr, d_odata[gpu], threads, blocks, f, mkt::cuda_streams[gpu], gpu);
				}
				mkt::sync_streams();
				
				// fold on gpus: step 2
				while(blocks > 1){
			      int threads_2 = blocks < 1024 ? blocks : 1024; // nextPow2
			      int blocks_2 = (blocks + threads_2 - 1) / threads_2;
				  for(int gpu = 0; gpu < 4; ++gpu){
				      hipSetDevice(gpu);
				      mkt::kernel::reduce_plus_call<double>(blocks, d_odata[gpu], d_odata[gpu], threads_2, blocks_2, mkt::cuda_streams[gpu], gpu);
				  }
				  blocks = blocks_2;
			  	  mkt::sync_streams();
			  	}
				
				// copy final sum from device to host
				  for (int gpu = 0; gpu < 4; ++gpu) {
				    hipSetDevice(gpu);
				    hipMemcpyAsync(&gpu_results[gpu], d_odata[gpu], sizeof(double), hipMemcpyDeviceToHost, mkt::cuda_streams[gpu]);
				  }
				  mkt::sync_streams();
				  
				  for(int gpu = 0; gpu < 4; ++gpu) {
					hipSetDevice(gpu);
					hipFree(d_odata[gpu]);
				  }
				
				for(int gpu = 0; gpu < 4; ++gpu){
					local_result = local_result + gpu_results[gpu];
				}
			}else if(a.get_device_distribution() == mkt::Distribution::COPY){ // use only gpu 0, since all have the same data
				const int gpu_elements = a.get_size_gpu();
				int threads = gpu_elements < 1024 ? gpu_elements : 1024; // nextPow2
				int blocks = (gpu_elements + threads - 1) / threads;
				hipSetDevice(0);
				double* d_odata;
				hipMalloc((void**) &d_odata, blocks * sizeof(double));
				Fish* devptr = a.get_device_pointer(0);
				
				mkt::kernel::map_reduce_plus_call<Fish, double, Lambda58_map_reduce_array_functor>(gpu_elements, devptr, d_odata, threads, blocks, f, mkt::cuda_streams[0], 0);
				
				// fold on gpus: step 2
				while(blocks > 1){
				  int threads_2 = blocks < 1024 ? blocks : 1024; // nextPow2
				  int blocks_2 = (blocks + threads_2 - 1) / threads_2;
				  mkt::kernel::reduce_plus_call<double>(blocks, d_odata, d_odata, threads_2, blocks_2, mkt::cuda_streams[0], 0);
				  blocks = blocks_2;
				}
				
				// copy final sum from device to host
				  hipMemcpyAsync(&local_result, d_odata, sizeof(double), hipMemcpyDeviceToHost, mkt::cuda_streams[0]);
				  mkt::sync_streams();
				hipFree(d_odata);
			}
			
			if(a.get_distribution() == mkt::Distribution::DIST){
				MPI_Allreduce(&local_result, &global_result, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
				return global_result;
			}else if(a.get_distribution() == mkt::Distribution::COPY){
				return local_result;
			}
	}
	template<>
	std::array<double,512> mkt::map_reduce_plus<Fish, std::array<double,512>, Lambda59_map_reduce_array_functor>(mkt::DArray<Fish>& a, Lambda59_map_reduce_array_functor f){
		std::array<double,512> local_result;
		local_result.fill(0.0);
		std::array<double,512> global_result;
		global_result.fill(0.0);
		
		if(a.get_device_distribution() == mkt::Distribution::DIST){
			for(int gpu = 0; gpu < 4; ++gpu){
				acc_set_device_num(gpu, acc_device_not_host);
				f.init(gpu);
				Fish* devptr = a.get_device_pointer(gpu);
				const int gpu_elements = a.get_size_gpu();
				std::array<double,512> gpu_result;
				gpu_result.fill(0.0);
				
				#pragma acc parallel loop deviceptr(devptr) present_or_copy(gpu_result) async(0)
				for(unsigned int counter = 0; counter < 512; ++counter) {
					double element_result = 0.0;
					#pragma acc loop reduction(+:element_result)
					for(unsigned int inner_counter = 0; inner_counter < gpu_elements; ++inner_counter) {
						double map_result = (f(devptr[inner_counter]))[counter]; // this is actually calculate more often than necessary
						element_result = element_result + map_result;
					}
					gpu_result[counter] = gpu_result[counter] + element_result;
				}
				acc_wait(0);
				
				for(unsigned int counter = 0; counter < 512; ++counter){
					local_result[counter] = local_result[counter] + gpu_result[counter];
				}
			}
		}else if(a.get_device_distribution() == mkt::Distribution::COPY){
			acc_set_device_num(0, acc_device_not_host);
			f.init(0);
			Fish* devptr = a.get_device_pointer(0);
			const int gpu_elements = a.get_size_gpu();
			
			#pragma acc parallel loop deviceptr(devptr) present_or_copy(local_result) async(0)
			for(unsigned int counter = 0; counter < 512; ++counter) {
				double element_result = 0.0;
				#pragma acc loop reduction(+:element_result)
				for(unsigned int inner_counter = 0; inner_counter < gpu_elements; ++inner_counter) {
					double map_result = (f(devptr[inner_counter]))[counter];
					element_result = element_result + map_result;
				}
				local_result[counter] = local_result[counter] + element_result;
			}
			acc_wait(0);
		}
		
		if(a.get_distribution() == mkt::Distribution::DIST){
			MPI_Allreduce(local_result.data(), global_result.data(), 512, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
			return global_result;
		}else if(a.get_distribution() == mkt::Distribution::COPY){
			return local_result;
		}				
		 // TODO
	}
	template<>
	double mkt::map_reduce_plus<Fish, double, Lambda60_map_reduce_array_functor>(mkt::DArray<Fish>& a, Lambda60_map_reduce_array_functor f){
			double local_result = 0.0;
			double global_result = 0.0;
						
			if(a.get_device_distribution() == mkt::Distribution::DIST){
				std::array<double*,4> d_odata;
				std::array<double, 4> gpu_results;
				const int gpu_elements = a.get_size_gpu();
				int threads = gpu_elements < 1024 ? gpu_elements : 1024; // nextPow2
				int blocks = (gpu_elements + threads - 1) / threads;
		
				for(int gpu = 0; gpu < 4; ++gpu){
					hipSetDevice(gpu);
					hipMalloc((void**) &d_odata[gpu], blocks * sizeof(double));
					Fish* devptr = a.get_device_pointer(gpu);
					
					mkt::kernel::map_reduce_plus_call<Fish, double, Lambda60_map_reduce_array_functor>(gpu_elements, devptr, d_odata[gpu], threads, blocks, f, mkt::cuda_streams[gpu], gpu);
				}
				mkt::sync_streams();
				
				// fold on gpus: step 2
				while(blocks > 1){
			      int threads_2 = blocks < 1024 ? blocks : 1024; // nextPow2
			      int blocks_2 = (blocks + threads_2 - 1) / threads_2;
				  for(int gpu = 0; gpu < 4; ++gpu){
				      hipSetDevice(gpu);
				      mkt::kernel::reduce_plus_call<double>(blocks, d_odata[gpu], d_odata[gpu], threads_2, blocks_2, mkt::cuda_streams[gpu], gpu);
				  }
				  blocks = blocks_2;
			  	  mkt::sync_streams();
			  	}
				
				// copy final sum from device to host
				  for (int gpu = 0; gpu < 4; ++gpu) {
				    hipSetDevice(gpu);
				    hipMemcpyAsync(&gpu_results[gpu], d_odata[gpu], sizeof(double), hipMemcpyDeviceToHost, mkt::cuda_streams[gpu]);
				  }
				  mkt::sync_streams();
				  
				  for(int gpu = 0; gpu < 4; ++gpu) {
					hipSetDevice(gpu);
					hipFree(d_odata[gpu]);
				  }
				
				for(int gpu = 0; gpu < 4; ++gpu){
					local_result = local_result + gpu_results[gpu];
				}
			}else if(a.get_device_distribution() == mkt::Distribution::COPY){ // use only gpu 0, since all have the same data
				const int gpu_elements = a.get_size_gpu();
				int threads = gpu_elements < 1024 ? gpu_elements : 1024; // nextPow2
				int blocks = (gpu_elements + threads - 1) / threads;
				hipSetDevice(0);
				double* d_odata;
				hipMalloc((void**) &d_odata, blocks * sizeof(double));
				Fish* devptr = a.get_device_pointer(0);
				
				mkt::kernel::map_reduce_plus_call<Fish, double, Lambda60_map_reduce_array_functor>(gpu_elements, devptr, d_odata, threads, blocks, f, mkt::cuda_streams[0], 0);
				
				// fold on gpus: step 2
				while(blocks > 1){
				  int threads_2 = blocks < 1024 ? blocks : 1024; // nextPow2
				  int blocks_2 = (blocks + threads_2 - 1) / threads_2;
				  mkt::kernel::reduce_plus_call<double>(blocks, d_odata, d_odata, threads_2, blocks_2, mkt::cuda_streams[0], 0);
				  blocks = blocks_2;
				}
				
				// copy final sum from device to host
				  hipMemcpyAsync(&local_result, d_odata, sizeof(double), hipMemcpyDeviceToHost, mkt::cuda_streams[0]);
				  mkt::sync_streams();
				hipFree(d_odata);
			}
			
			if(a.get_distribution() == mkt::Distribution::DIST){
				MPI_Allreduce(&local_result, &global_result, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
				return global_result;
			}else if(a.get_distribution() == mkt::Distribution::COPY){
				return local_result;
			}
	}
	template<>
	std::array<double,512> mkt::map_reduce_plus<Fish, std::array<double,512>, Lambda61_map_reduce_array_functor>(mkt::DArray<Fish>& a, Lambda61_map_reduce_array_functor f){
		std::array<double,512> local_result;
		local_result.fill(0.0);
		std::array<double,512> global_result;
		global_result.fill(0.0);
		
		if(a.get_device_distribution() == mkt::Distribution::DIST){
			for(int gpu = 0; gpu < 4; ++gpu){
				acc_set_device_num(gpu, acc_device_not_host);
				f.init(gpu);
				Fish* devptr = a.get_device_pointer(gpu);
				const int gpu_elements = a.get_size_gpu();
				std::array<double,512> gpu_result;
				gpu_result.fill(0.0);
				
				#pragma acc parallel loop deviceptr(devptr) present_or_copy(gpu_result) async(0)
				for(unsigned int counter = 0; counter < 512; ++counter) {
					double element_result = 0.0;
					#pragma acc loop reduction(+:element_result)
					for(unsigned int inner_counter = 0; inner_counter < gpu_elements; ++inner_counter) {
						double map_result = (f(devptr[inner_counter]))[counter]; // this is actually calculate more often than necessary
						element_result = element_result + map_result;
					}
					gpu_result[counter] = gpu_result[counter] + element_result;
				}
				acc_wait(0);
				
				for(unsigned int counter = 0; counter < 512; ++counter){
					local_result[counter] = local_result[counter] + gpu_result[counter];
				}
			}
		}else if(a.get_device_distribution() == mkt::Distribution::COPY){
			acc_set_device_num(0, acc_device_not_host);
			f.init(0);
			Fish* devptr = a.get_device_pointer(0);
			const int gpu_elements = a.get_size_gpu();
			
			#pragma acc parallel loop deviceptr(devptr) present_or_copy(local_result) async(0)
			for(unsigned int counter = 0; counter < 512; ++counter) {
				double element_result = 0.0;
				#pragma acc loop reduction(+:element_result)
				for(unsigned int inner_counter = 0; inner_counter < gpu_elements; ++inner_counter) {
					double map_result = (f(devptr[inner_counter]))[counter];
					element_result = element_result + map_result;
				}
				local_result[counter] = local_result[counter] + element_result;
			}
			acc_wait(0);
		}
		
		if(a.get_distribution() == mkt::Distribution::DIST){
			MPI_Allreduce(local_result.data(), global_result.data(), 512, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
			return global_result;
		}else if(a.get_distribution() == mkt::Distribution::COPY){
			return local_result;
		}				
		 // TODO
	}
	template<>
	double mkt::map_reduce_max<Fish, double, Lambda62_map_reduce_array_functor>(mkt::DArray<Fish>& a, Lambda62_map_reduce_array_functor f){
			double local_result = std::numeric_limits<double>::lowest();
			double global_result = std::numeric_limits<double>::lowest();
						
			if(a.get_device_distribution() == mkt::Distribution::DIST){
				std::array<double*,4> d_odata;
				std::array<double, 4> gpu_results;
				const int gpu_elements = a.get_size_gpu();
				int threads = gpu_elements < 1024 ? gpu_elements : 1024; // nextPow2
				int blocks = (gpu_elements + threads - 1) / threads;
		
				for(int gpu = 0; gpu < 4; ++gpu){
					hipSetDevice(gpu);
					hipMalloc((void**) &d_odata[gpu], blocks * sizeof(double));
					Fish* devptr = a.get_device_pointer(gpu);
					
					mkt::kernel::map_reduce_max_call<Fish, double, Lambda62_map_reduce_array_functor>(gpu_elements, devptr, d_odata[gpu], threads, blocks, f, mkt::cuda_streams[gpu], gpu);
				}
				mkt::sync_streams();
				
				// fold on gpus: step 2
				while(blocks > 1){
			      int threads_2 = blocks < 1024 ? blocks : 1024; // nextPow2
			      int blocks_2 = (blocks + threads_2 - 1) / threads_2;
				  for(int gpu = 0; gpu < 4; ++gpu){
				      hipSetDevice(gpu);
				      mkt::kernel::reduce_max_call<double>(blocks, d_odata[gpu], d_odata[gpu], threads_2, blocks_2, mkt::cuda_streams[gpu], gpu);
				  }
				  blocks = blocks_2;
			  	  mkt::sync_streams();
			  	}
				
				// copy final sum from device to host
				  for (int gpu = 0; gpu < 4; ++gpu) {
				    hipSetDevice(gpu);
				    hipMemcpyAsync(&gpu_results[gpu], d_odata[gpu], sizeof(double), hipMemcpyDeviceToHost, mkt::cuda_streams[gpu]);
				  }
				  mkt::sync_streams();
				  
				  for(int gpu = 0; gpu < 4; ++gpu) {
					hipSetDevice(gpu);
					hipFree(d_odata[gpu]);
				  }
				
				for(int gpu = 0; gpu < 4; ++gpu){
					local_result = local_result > gpu_results[gpu] ? local_result : gpu_results[gpu];
				}
			}else if(a.get_device_distribution() == mkt::Distribution::COPY){ // use only gpu 0, since all have the same data
				const int gpu_elements = a.get_size_gpu();
				int threads = gpu_elements < 1024 ? gpu_elements : 1024; // nextPow2
				int blocks = (gpu_elements + threads - 1) / threads;
				hipSetDevice(0);
				double* d_odata;
				hipMalloc((void**) &d_odata, blocks * sizeof(double));
				Fish* devptr = a.get_device_pointer(0);
				
				mkt::kernel::map_reduce_max_call<Fish, double, Lambda62_map_reduce_array_functor>(gpu_elements, devptr, d_odata, threads, blocks, f, mkt::cuda_streams[0], 0);
				
				// fold on gpus: step 2
				while(blocks > 1){
				  int threads_2 = blocks < 1024 ? blocks : 1024; // nextPow2
				  int blocks_2 = (blocks + threads_2 - 1) / threads_2;
				  mkt::kernel::reduce_max_call<double>(blocks, d_odata, d_odata, threads_2, blocks_2, mkt::cuda_streams[0], 0);
				  blocks = blocks_2;
				}
				
				// copy final sum from device to host
				  hipMemcpyAsync(&local_result, d_odata, sizeof(double), hipMemcpyDeviceToHost, mkt::cuda_streams[0]);
				  mkt::sync_streams();
				hipFree(d_odata);
			}
			
			if(a.get_distribution() == mkt::Distribution::DIST){
				MPI_Allreduce(&local_result, &global_result, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
				return global_result;
			}else if(a.get_distribution() == mkt::Distribution::COPY){
				return local_result;
			}
	}
	
	
	
	int main(int argc, char** argv) {
		MPI_Init(&argc, &argv);
		
		MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);
		MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
		
		if(mpi_world_size != number_of_processes || mpi_rank != process_id){
			MPI_Finalize();
			return EXIT_FAILURE;
		}				
		mkt::init();
		
		
		mkt::sync_streams();
		
		mkt::DArray<Fish> population(12, 2048, 128, Fish{}, 4, 12, 1536, mkt::DIST, mkt::COPY);
		mkt::DArray<double> instinctive_movement_vector_copy(12, 512, 512, 0.0, 1, 12, 0, mkt::COPY, mkt::COPY);
		mkt::DArray<Fish> weighted_fishes(12, 2048, 128, Fish{}, 4, 12, 1536, mkt::DIST, mkt::COPY);
		mkt::DArray<double> barycenter_copy(12, 512, 512, 0.0, 1, 12, 0, mkt::COPY, mkt::COPY);
		
		InitFish_map_in_place_array_functor initFish_map_in_place_array_functor{};
		EvaluateFitness_map_in_place_array_functor evaluateFitness_map_in_place_array_functor{};
		IndividualMovement_map_in_place_array_functor individualMovement_map_in_place_array_functor{};
		Feeding_map_in_place_array_functor feeding_map_in_place_array_functor{};
		CalcDisplacementMap_map_in_place_array_functor calcDisplacementMap_map_in_place_array_functor{};
		CalcInstinctiveMovementVector_map_in_place_array_functor calcInstinctiveMovementVector_map_in_place_array_functor{};
		InstinctiveMovement_map_in_place_array_functor instinctiveMovement_map_in_place_array_functor{instinctive_movement_vector_copy};
		CalcWeightedFish_map_array_functor calcWeightedFish_map_array_functor{};
		CalcBarycenterMap_map_in_place_array_functor calcBarycenterMap_map_in_place_array_functor{};
		VolitiveMovement_map_in_place_array_functor volitiveMovement_map_in_place_array_functor{barycenter_copy};
		Lambda56_map_reduce_array_functor lambda56_map_reduce_array_functor{};
		Lambda57_map_reduce_array_functor lambda57_map_reduce_array_functor{};
		Lambda58_map_reduce_array_functor lambda58_map_reduce_array_functor{};
		Lambda59_map_reduce_array_functor lambda59_map_reduce_array_functor{};
		Lambda60_map_reduce_array_functor lambda60_map_reduce_array_functor{};
		Lambda61_map_reduce_array_functor lambda61_map_reduce_array_functor{};
		Lambda62_map_reduce_array_functor lambda62_map_reduce_array_functor{};
		
		
				
			MPI_Datatype Fish_mpi_type_temp;
			MPI_Type_create_struct(9, (std::array<int,9>{1, 1, 1, 1, 1, 1, 1, 1, 1}).data(), (std::array<MPI_Aint,9>{static_cast<MPI_Aint>(offsetof(struct Fish, position)), static_cast<MPI_Aint>(offsetof(struct Fish, fitness)), static_cast<MPI_Aint>(offsetof(struct Fish, candidate_position)), static_cast<MPI_Aint>(offsetof(struct Fish, candidate_fitness)), static_cast<MPI_Aint>(offsetof(struct Fish, displacement)), static_cast<MPI_Aint>(offsetof(struct Fish, fitness_variation)), static_cast<MPI_Aint>(offsetof(struct Fish, weight)), static_cast<MPI_Aint>(offsetof(struct Fish, best_position)), static_cast<MPI_Aint>(offsetof(struct Fish, best_fitness))}).data(), (std::array<MPI_Datatype,9>{MPI_DOUBLE, MPI_DOUBLE, MPI_DOUBLE, MPI_DOUBLE, MPI_DOUBLE, MPI_DOUBLE, MPI_DOUBLE, MPI_DOUBLE, MPI_DOUBLE}).data(), &Fish_mpi_type_temp);
			MPI_Type_create_resized(Fish_mpi_type_temp, 0, sizeof(Fish), &Fish_mpi_type);
			MPI_Type_free(&Fish_mpi_type_temp);
			MPI_Type_commit(&Fish_mpi_type);
			
			
		
			
		
		
		mkt::sync_streams();
		mkt::map_in_place<Fish, InitFish_map_in_place_array_functor>(population, initFish_map_in_place_array_functor);
		double step_size = (STEP_SIZE_INITIAL);
		double step_size_vol = (STEP_SIZE_VOLITIVE_INITIAL);
		double sum_weight_last_iteration = 0.0;
		sum_weight_last_iteration = mkt::map_reduce_plus<Fish, double, Lambda56_map_reduce_array_functor>(population, lambda56_map_reduce_array_functor);
		for(int iteration = 0; ((iteration) < (ITERATIONS)); ++iteration){
			mkt::map_in_place<Fish, EvaluateFitness_map_in_place_array_functor>(population, evaluateFitness_map_in_place_array_functor);
			if(((iteration) > 0)){
				step_size = ((step_size) - (((STEP_SIZE_INITIAL) - (STEP_SIZE_FINAL)) / static_cast<double>(((ITERATIONS) - 1))));
				step_size_vol = ((step_size_vol) - (((STEP_SIZE_VOLITIVE_INITIAL) - (STEP_SIZE_VOLITIVE_FINAL)) / static_cast<double>(((ITERATIONS) - 1))));
			}
			individualMovement_map_in_place_array_functor.step_size = (step_size);
			mkt::map_in_place<Fish, IndividualMovement_map_in_place_array_functor>(population, individualMovement_map_in_place_array_functor);
			double max_fitness_variation = 0.0;
			max_fitness_variation = mkt::map_reduce_max<Fish, double, Lambda57_map_reduce_array_functor>(population, lambda57_map_reduce_array_functor);
			feeding_map_in_place_array_functor.max_fitness_variation = (max_fitness_variation);
			mkt::map_in_place<Fish, Feeding_map_in_place_array_functor>(population, feeding_map_in_place_array_functor);
			double sum_fitness_variation = 0.0;
			sum_fitness_variation = mkt::map_reduce_plus<Fish, double, Lambda58_map_reduce_array_functor>(population, lambda58_map_reduce_array_functor);
			mkt::map_in_place<Fish, CalcDisplacementMap_map_in_place_array_functor>(population, calcDisplacementMap_map_in_place_array_functor);
			instinctive_movement_vector_copy = mkt::map_reduce_plus<Fish, std::array<double,512>, Lambda59_map_reduce_array_functor>(population, lambda59_map_reduce_array_functor);
			calcInstinctiveMovementVector_map_in_place_array_functor.sum_fitness_variation = (sum_fitness_variation);
			mkt::map_in_place<double, CalcInstinctiveMovementVector_map_in_place_array_functor>(instinctive_movement_vector_copy, calcInstinctiveMovementVector_map_in_place_array_functor);
			mkt::map_in_place<Fish, InstinctiveMovement_map_in_place_array_functor>(population, instinctiveMovement_map_in_place_array_functor);
			double sum_weight = 0.0;
			sum_weight = mkt::map_reduce_plus<Fish, double, Lambda60_map_reduce_array_functor>(population, lambda60_map_reduce_array_functor);
			mkt::map<Fish, Fish, CalcWeightedFish_map_array_functor>(population, weighted_fishes, calcWeightedFish_map_array_functor);
			barycenter_copy = mkt::map_reduce_plus<Fish, std::array<double,512>, Lambda61_map_reduce_array_functor>(weighted_fishes, lambda61_map_reduce_array_functor);
			calcBarycenterMap_map_in_place_array_functor.sum_weight = (sum_weight);
			mkt::map_in_place<double, CalcBarycenterMap_map_in_place_array_functor>(barycenter_copy, calcBarycenterMap_map_in_place_array_functor);
			volitiveMovement_map_in_place_array_functor.step_size = (step_size_vol);volitiveMovement_map_in_place_array_functor.sum_weight = (sum_weight);volitiveMovement_map_in_place_array_functor.sum_weight_last_iteration = (sum_weight_last_iteration);
			mkt::map_in_place<Fish, VolitiveMovement_map_in_place_array_functor>(population, volitiveMovement_map_in_place_array_functor);
			sum_weight_last_iteration = (sum_weight);
		}
		double global_best_fitness = 0.0;
		global_best_fitness = mkt::map_reduce_max<Fish, double, Lambda62_map_reduce_array_functor>(population, lambda62_map_reduce_array_functor);
		mkt::sync_streams();
		
		mkt::sync_streams();
		
		
		MPI_Finalize();
		return EXIT_SUCCESS;
		}
